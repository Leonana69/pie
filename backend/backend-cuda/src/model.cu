#include "hip/hip_runtime.h"
#include "model.hpp"

// All implementation-specific headers are safely included here
#include "l4ma.cuh"
#include "ztensor.hpp"
#include "common.cuh"
#include "stack_allocator.cuh"
#include <iostream>
#include <set>
#include <memory>
#include <stdexcept>
#include <hip/hip_runtime.h>
#include <vector>
#include <map>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>


// --- Internal data structures ---

// Represents a block in the KV cache on the CPU.
struct Block {
    std::vector<uint32_t> position_ids;
    std::vector<bool> occupancy;

    Block() = default; // Default constructor
    Block(int32_t kv_page_size)
        : position_ids(kv_page_size, 0), occupancy(kv_page_size, false) {}
};

// Represents a text embedding on the CPU.
struct TextEmbed {
    uint32_t token_id;
    uint32_t position_id;
};


// The actual implementation of the Server is hidden in this struct.
struct Model::ModelImpl {
    std::unique_ptr<L4maForCausalLM<__hip_bfloat16>> model;

    // --- State Management ---
    std::map<uint32_t, Block> blocks;
    std::map<uint32_t, TextEmbed> embeds;

    // Storage for results of embedding/inference, analogous to Python's embed_storage
    thrust::device_vector<__hip_bfloat16> embed_storage_p1;
    thrust::device_vector<int32_t> embed_storage_p2;

    // Configuration
    int32_t kv_page_size;
    int32_t dist_size;

    // --- Handler method declarations added to ModelImpl ---
    // These methods contain the core logic and have access to the model pointer.
    void handle_allocate(const std::vector<Model::AllocateCommand>& commands);
    void handle_deallocate(const std::vector<Model::DeallocateCommand>& commands);
    void handle_embed_text(const std::vector<Model::EmbedTextCommand>& commands);
    void handle_fill_block(const std::vector<Model::FillBlockCommand>& commands);
    void handle_mask_block(const std::vector<Model::MaskBlockCommand>& commands);
    void handle_copy_block(const std::vector<Model::CopyBlockCommand>& commands);
    void handle_decode_token_distribution(const std::vector<Model::DecodeTokenDistributionCommand>& commands);
    std::vector<Model::SampleTopKResult> handle_sample_top_k(const std::vector<Model::SampleTopKCommand>& commands);
};

namespace { 

template<typename T>
std::unique_ptr<L4maForCausalLM<T>> load_model_internal(const AppConfig& config, const ModelMetadata& metadata) {
    std::cout << "Instantiating model structure on device..." << std::endl;

    auto model_ptr = std::make_unique<L4maForCausalLM<T>>(metadata.architecture);

    auto params_map = model_ptr->get_parameters();
    std::cout << "Found " << params_map.size() << " parameter tensors in the model structure." << std::endl;

    const auto model_dir = config.cache_dir / config.model_name;
    std::set<std::string> loaded_keys;

    for (const auto& param_file : metadata.parameters) {
        std::filesystem::path weights_path = model_dir / param_file;
        std::cout << "Reading weights from: " << weights_path.string() << std::endl;

        try {
            ztensor::zTensorReader reader(weights_path.string());
            for (const auto& name : reader.list_tensors()) {
                if (params_map.count(name) && !loaded_keys.count(name)) {
                    const auto& info = reader.get_tensor_info(name);
                    auto& target_tensor_ptr = params_map[name];

                    if (target_tensor_ptr->size() != info.num_elements()) {
                        std::cerr << "    Warning: Shape mismatch for tensor '" << name << "'. ZT: " << info.num_elements() << ", Model: " << target_tensor_ptr->size() << ". Skipping." << std::endl;
                        continue;
                    }

                    const T* host_ptr = static_cast<const T*>(reader.get_raw_tensor_pointer(name));
                    if (host_ptr) {
                        hipMemcpy(thrust::raw_pointer_cast(target_tensor_ptr->data()), host_ptr, info.size, hipMemcpyHostToDevice);
                        loaded_keys.insert(name);
                    }
                }
            }
        } catch (const std::runtime_error& e) {
            std::cerr << "Warning: Could not read file " << weights_path.string() << ". Error: " << e.what() << std::endl;
        }
    }

    if (params_map.count("lm_head.weight") && params_map.count("model.embed_tokens.weight")) {
        params_map["lm_head.weight"] = params_map["model.embed_tokens.weight"];
        loaded_keys.insert("lm_head.weight");
    }
    
    if (loaded_keys.size() != params_map.size()) {
        std::cout << "\nWarning: Mismatch between loaded and expected parameter counts." << std::endl;
        std::cout << "Missing parameters:" << std::endl;
        for (const auto& pair : params_map) {
            if (loaded_keys.find(pair.first) == loaded_keys.end()) {
                std::cout << "  - " << pair.first << std::endl;
            }
        }
    }
    
    std::cout << "\nSuccessfully loaded " << loaded_keys.size() << " expected weights." << std::endl;

    return model_ptr;
}

} // anonymous namespace

// --- New: Placeholder implementations for handler methods ---
// These are the actual implementations within the ModelImpl struct.

void Model::ModelImpl::handle_allocate(const std::vector<Model::AllocateCommand>& commands) {
    std::cout << "  [ModelImpl] handle_allocate called with " << commands.size() << " items." << std::endl;
    for (const auto& cmd : commands) {
        if (cmd.kind == Model::ObjectKind::KV_BLOCK) {
            for (uint32_t i = 0; i < cmd.count; ++i) {
                uint32_t block_id = cmd.object_id_offset + i;
                blocks[block_id] = Block(kv_page_size);
            }
        }
    }
}

void Model::ModelImpl::handle_deallocate(const std::vector<Model::DeallocateCommand>& commands) {
    std::cout << "  [ModelImpl] handle_deallocate called with " << commands.size() << " items." << std::endl;
    // Currently a no-op, as in the Python implementation.
    // Blocks are cleared implicitly when the model is destroyed.
}

void Model::ModelImpl::handle_embed_text(const std::vector<Model::EmbedTextCommand>& commands) {
    std::cout << "  [ModelImpl] handle_embed_text called with " << commands.size() << " items." << std::endl;
    for (const auto& cmd : commands) {
        embeds[cmd.embedding_id] = {cmd.token_id, cmd.position_id};
    }
}
// Assuming other necessary includes and the Model::ModelImpl class definition exist above

void Model::ModelImpl::handle_fill_block(const std::vector<Model::FillBlockCommand>& commands) {

    std::cout << "  [ModelImpl] handle_fill_block called with " << commands.size() << " items." << std::endl;

    // --- Host-side vector preparations ---
    std::vector<int32_t> kv_page_indices_host;
    std::vector<int32_t> kv_page_indptr_host = {0};
    std::vector<int32_t> kv_last_page_lens_host;
    std::vector<int32_t> qo_indptr_host = {0};
    std::vector<uint8_t> custom_masks_host;
    std::vector<int32_t> mask_indptr_host = {0};
    std::vector<int32_t> kv_batch_indices_host;
    std::vector<int32_t> kv_positions_host;
    std::vector<uint32_t> new_token_ids_host;
    std::vector<int32_t> new_position_ids_host;

    struct OutputEmbedPostproc {
        size_t logit_row_idx;
        uint32_t dest_embed_id;
    };
    std::vector<OutputEmbedPostproc> output_embed_postproc;

    int batch_idx = 0;
    for (const auto& cmd : commands) {
        kv_page_indices_host.insert(kv_page_indices_host.end(), cmd.context_block_ids.begin(), cmd.context_block_ids.end());
        kv_page_indptr_host.push_back(kv_page_indices_host.size());
        kv_last_page_lens_host.push_back(cmd.last_block_len);

        int32_t num_new_tokens = cmd.input_embedding_ids.size();
        qo_indptr_host.push_back(qo_indptr_host.back() + num_new_tokens);

        size_t total_ctx_tokens = (cmd.context_block_ids.empty()) ? 0 :
                                  kv_page_size * (cmd.context_block_ids.size() - 1) + cmd.last_block_len;

        mask_indptr_host.push_back(mask_indptr_host.back() + (num_new_tokens * total_ctx_tokens));

        for (int32_t i = 0; i < num_new_tokens; ++i) {
            kv_batch_indices_host.push_back(batch_idx);
            kv_positions_host.push_back(total_ctx_tokens - cmd.last_block_len + i);
        }

        std::vector<uint32_t> inp_pos_ids_for_mask;

        for (size_t i = 0; i < cmd.input_embedding_ids.size(); ++i) {
            uint32_t embed_id = cmd.input_embedding_ids[i];
            auto it = embeds.find(embed_id);
            if (it != embeds.end()) {
                const auto& embed = it->second;
                new_token_ids_host.push_back(embed.token_id);
                new_position_ids_host.push_back(embed.position_id);
                inp_pos_ids_for_mask.push_back(embed.position_id);

                size_t token_abs_pos = total_ctx_tokens - num_new_tokens + i;
                uint32_t tgt_block_idx = token_abs_pos / kv_page_size;
                uint32_t tgt_block_offset = token_abs_pos % kv_page_size;

                // print tgt_block_idx and tgt_block_offset for debugging
                std::cout << "Processing token: " << embed.token_id 
                          << ", position: " << embed.position_id 
                          << ", token_abs_pos: " << token_abs_pos
                          << ", target block index: " << tgt_block_idx 
                          << ", target block offset: " << tgt_block_offset << std::endl;

                if (tgt_block_idx < cmd.context_block_ids.size()) {
                    uint32_t tgt_block_id = cmd.context_block_ids[tgt_block_idx];
                    auto block_it = blocks.find(tgt_block_id);
                    if (block_it != blocks.end()) {
                        block_it->second.occupancy[tgt_block_offset] = true;
                        block_it->second.position_ids[tgt_block_offset] = embed.position_id;
                    }
                }
            }
        }

        for (size_t i = 0; i < cmd.output_embedding_ids.size(); ++i) {
            size_t logit_row = new_token_ids_host.size() - cmd.output_embedding_ids.size() + i;
            output_embed_postproc.push_back({logit_row, cmd.output_embedding_ids[i]});
        }

        if (total_ctx_tokens > 0) {
            std::vector<uint32_t> ctx_pos_ids;
            std::vector<bool> ctx_occupancy;
            ctx_pos_ids.reserve(total_ctx_tokens);
            ctx_occupancy.reserve(total_ctx_tokens);

            for (size_t i = 0; i < cmd.context_block_ids.size(); ++i) {
                uint32_t block_id = cmd.context_block_ids[i];
                const auto& block = blocks.at(block_id);
                size_t len_to_copy = (i == cmd.context_block_ids.size() - 1) ? cmd.last_block_len : kv_page_size;
                ctx_pos_ids.insert(ctx_pos_ids.end(), block.position_ids.begin(), block.position_ids.begin() + len_to_copy);
                ctx_occupancy.insert(ctx_occupancy.end(), block.occupancy.begin(), block.occupancy.begin() + len_to_copy);
            }

            // print all ctx_pos_ids and ctx_occupancy for debugging
            std::cout << "ctx_pos_ids: ";
            for (const auto& pos_id : ctx_pos_ids) {
                std::cout << pos_id << " ";
            }
            std::cout << "\nctx_occupancy: ";
            for (const auto& occ : ctx_occupancy) {
                std::cout << (occ ? 1 : 0) << " ";
            }
            std::cout << std::endl;


            for (uint32_t inp_pos_id : inp_pos_ids_for_mask) {
                for (size_t j = 0; j < total_ctx_tokens; ++j) {
                    bool causal_mask = ctx_pos_ids[j] <= inp_pos_id;
                    bool valid_mask = ctx_occupancy[j];
                    custom_masks_host.push_back((causal_mask && valid_mask) ? 1 : 0);
                }
            }
        }
        batch_idx++;
    }

    // print all host vectors for debugging
    std::cout << "kv_page_indices_host: ";
    for (const auto& idx : kv_page_indices_host) {
        std::cout << idx << " ";
    }
    std::cout << "\nkv_page_indptr_host: ";
    for (const auto& idx : kv_page_indptr_host) {
        std::cout << idx << " ";
    }
    std::cout << "\nkv_last_page_lens_host: ";
    for (const auto& len : kv_last_page_lens_host) {
        std::cout << len << " ";
    }
    std::cout << "\nqo_indptr_host: ";
    for (const auto& idx : qo_indptr_host) {
        std::cout << idx << " ";
    }
    std::cout << "\ncustom_masks_host: ";
    for (const auto& mask : custom_masks_host) {
        std::cout << static_cast<int>(mask) << " ";
    }
    std::cout << "\nmask_indptr_host: ";
    for (const auto& idx : mask_indptr_host) {
        std::cout << idx << " ";
    }
    std::cout << "\nnew_token_ids_host: ";
    for (const auto& token_id : new_token_ids_host) {
        std::cout << token_id << " ";
    }
    std::cout << "\nnew_position_ids_host: ";
    for (const auto& pos_id : new_position_ids_host) {
        std::cout << pos_id << " ";
    }
    std::cout << "\nkv_batch_indices_host: ";
    for (const auto& batch_idx : kv_batch_indices_host) {
        std::cout << batch_idx << " ";
    }
    std::cout << "\nkv_positions_host: ";
    for (const auto& pos : kv_positions_host) {
        std::cout << pos << " ";
    }



    // --- Copy data to device ---
    thrust::device_vector<int32_t> kv_page_indices = kv_page_indices_host;
    thrust::device_vector<int32_t> kv_page_indptr = kv_page_indptr_host;
    thrust::device_vector<int32_t> kv_last_page_lens = kv_last_page_lens_host;
    thrust::device_vector<int32_t> qo_indptr = qo_indptr_host;
    thrust::device_vector<uint8_t> custom_mask = custom_masks_host;
    thrust::device_vector<int32_t> mask_indptr = mask_indptr_host;
    thrust::device_vector<uint32_t> new_token_ids = new_token_ids_host;
    thrust::device_vector<int32_t> new_position_ids = new_position_ids_host;
    thrust::device_vector<int32_t> kv_batch_indices = kv_batch_indices_host;
    thrust::device_vector<int32_t> kv_positions = kv_positions_host;

    // --- Allocate buffers ---
    size_t num_total_new_tokens = new_token_ids.size();
    if (num_total_new_tokens == 0) return;

    thrust::device_vector<__hip_bfloat16> logits(num_total_new_tokens * model->get_config().vocab_size);
    
    size_t workspace_size_bytes = model->get_workspace_size(num_total_new_tokens);
    thrust::device_vector<char> workspace_buffer(workspace_size_bytes);

    StackAllocator allocator(thrust::raw_pointer_cast(workspace_buffer.data()), workspace_size_bytes);

    hipStream_t stream = 0;

    // --- Model Forward Pass ---
    model->forward(
        allocator,
        thrust::raw_pointer_cast(logits.data()),
        new_token_ids,
        new_position_ids,
        kv_page_indices,
        kv_page_indptr,
        kv_page_indptr_host,
        kv_last_page_lens,
        qo_indptr,
        qo_indptr_host,
        custom_mask,
        mask_indptr,
        stream,
        kv_page_size,
        kv_batch_indices,
        kv_positions
    );

    // --- Post-processing ---
    if (!output_embed_postproc.empty()) {
        std::vector<size_t> logit_indices_host;
        std::vector<uint32_t> dest_embed_ids_host;
        logit_indices_host.reserve(output_embed_postproc.size());
        dest_embed_ids_host.reserve(output_embed_postproc.size());
        for (const auto& p : output_embed_postproc) {
            logit_indices_host.push_back(p.logit_row_idx);
            dest_embed_ids_host.push_back(p.dest_embed_id);
        }
        thrust::device_vector<size_t> logit_indices_dev = logit_indices_host;
        thrust::device_vector<uint32_t> dest_embed_ids_dev = dest_embed_ids_host;

        topk_scatter(
            logits,
            logit_indices_dev,
            dest_embed_ids_dev,
            model->get_config().vocab_size,
            dist_size,
            embed_storage_p1,
            embed_storage_p2,
            stream
        );
    }
}

void Model::ModelImpl::handle_mask_block(const std::vector<Model::MaskBlockCommand>& commands) {
    std::cout << "  [ModelImpl] handle_mask_block called with " << commands.size() << " items." << std::endl;
    for (const auto& cmd : commands) {
        auto it = blocks.find(cmd.block_id);
        if (it != blocks.end()) {
            Block& block = it->second;
            if (block.occupancy.size() == cmd.mask.size()) {
                block.occupancy = cmd.mask;
            } else {
                std::cerr << "Warning: Mask size mismatch for block " << cmd.block_id << std::endl;
            }
        } else {
            std::cerr << "Warning: Block not found for masking: " << cmd.block_id << std::endl;
        }
    }
}

void Model::ModelImpl::handle_copy_block(const std::vector<Model::CopyBlockCommand>& commands) {
    std::cout << "  [ModelImpl] handle_copy_block called with " << commands.size() << " items." << std::endl;
    // TODO: Implement hipMemcpy between different KV cache pages on the device.
    // This requires getting raw pointers to the device vectors for each layer in the KV cache.
}

void Model::ModelImpl::handle_decode_token_distribution(const std::vector<Model::DecodeTokenDistributionCommand>& commands) {
    std::cout << "  [ModelImpl] handle_decode_token_distribution called with " << commands.size() << " items." << std::endl;
    // This is a no-op in the provided python implementation.
    // The logic is integrated into fill_block where top-k results are computed and stored directly.
}

std::vector<Model::SampleTopKResult> Model::ModelImpl::handle_sample_top_k(const std::vector<Model::SampleTopKCommand>& commands) {
    std::cout << "  [ModelImpl] handle_sample_top_k called with " << commands.size() << " items." << std::endl;
    std::vector<Model::SampleTopKResult> results;
    results.reserve(commands.size());

    for (const auto& cmd : commands) {
        Model::SampleTopKResult res;
        
        // Determine the number of elements to copy
        uint32_t k = (cmd.k > 0 && cmd.k < static_cast<uint32_t>(dist_size)) ? cmd.k : dist_size;

        // Create host vectors to hold the results
        thrust::host_vector<__hip_bfloat16> topk_probs_host(k);
        thrust::host_vector<int32_t> topk_tokens_host(k);

        // Copy data from device to host
        hipMemcpy(topk_probs_host.data(), thrust::raw_pointer_cast(embed_storage_p1.data()) + cmd.distribution_id * dist_size, k * sizeof(__hip_bfloat16), hipMemcpyDeviceToHost);
        hipMemcpy(topk_tokens_host.data(), thrust::raw_pointer_cast(embed_storage_p2.data()) + cmd.distribution_id * dist_size, k * sizeof(int32_t), hipMemcpyDeviceToHost);
        
        res.token_ids.assign(topk_tokens_host.begin(), topk_tokens_host.end());
        
        res.probabilities.resize(k);
        for(size_t i = 0; i < k; ++i) {
            res.probabilities[i] = static_cast<float>(topk_probs_host[i]);
        }

        results.push_back(res);
    }
    return results;
}

// --- Public Interface Implementation ---

Model::Model(const AppConfig& config,const ModelMetadata& out_metadata)
    : pimpl(std::make_unique<ModelImpl>()) {
    
    std::cout << "Starting service..." << std::endl;
    // Load the model and store it in the implementation object
    pimpl->model = load_model_internal<__hip_bfloat16>(config, out_metadata);
    std::cout << "Model loaded successfully and is resident on the GPU." << std::endl;

    // initialize kv cache
    pimpl->model->create_kv_device_vectors(config.max_num_kv_pages);

    // Initialize state
    pimpl->kv_page_size = config.kv_page_size;
    pimpl->dist_size = config.dist_size;
    pimpl->embed_storage_p1.resize(config.max_num_embeds * config.dist_size);
    pimpl->embed_storage_p2.resize(config.max_num_embeds * config.dist_size);
}

Model::~Model() = default;

void Model::run() {
    // This function is now used as a test routine for handle_fill_block.

    std::cout << "\n--- [START] Running Test Routine for handle_fill_block ---" << std::endl;

    // 1. Define test parameters: a random sequence of tokens and IDs.
    const std::vector<uint32_t> token_ids = {3513, 5331, 533, 11};
    const uint32_t block_id = 101; // A unique ID for our KV block
    const uint32_t embed_id_offset = 201; // Starting ID for our input embeddings
    const uint32_t dist_id = 301;         // ID for the output distribution object

    // Ensure the tokens fit within a single page.
    if (token_ids.size() > static_cast<size_t>(pimpl->kv_page_size)) {
        std::cerr << "Test Error: Number of tokens exceeds kv_page_size." << std::endl;
        return;
    }

    // 2. Call handle_allocate to allocate a page for the KV cache.
    std::cout << "\n[Step 1] Allocating KV Block..." << std::endl;
    Model::AllocateCommand alloc_cmd;
    alloc_cmd.kind = Model::ObjectKind::KV_BLOCK;
    alloc_cmd.object_id_offset = block_id;
    alloc_cmd.count = 1;
    handle_allocate({alloc_cmd});
    std::cout << "Allocated block with ID: " << block_id << std::endl;

    // 3. Call handle_embed_texts to create mappings for token and position IDs.
    std::cout << "\n[Step 2] Creating Text Embeddings..." << std::endl;
    std::vector<Model::EmbedTextCommand> embed_cmds;
    std::vector<uint32_t> input_embed_ids;
    for (size_t i = 0; i < token_ids.size(); ++i) {
        uint32_t current_embed_id = embed_id_offset + i;
        input_embed_ids.push_back(current_embed_id);

        Model::EmbedTextCommand embed_cmd;
        embed_cmd.embedding_id = current_embed_id;
        embed_cmd.token_id = token_ids[i];
        embed_cmd.position_id = i; // Simple sequential positions 0, 1, 2, ...
        embed_cmds.push_back(embed_cmd);
    }
    handle_embed_text(embed_cmds);
    std::cout << "Created " << embed_cmds.size() << " embeddings." << std::endl;

    // 4. Call handle_fill_block to do a single forward pass.
    std::cout << "\n[Step 3] Calling handle_fill_block for a forward pass..." << std::endl;
    Model::FillBlockCommand fill_cmd;
    fill_cmd.last_block_len = token_ids.size(); // No previous context in the block
    fill_cmd.context_block_ids = {block_id}; // The block to fill with new KV data
    fill_cmd.input_embedding_ids = input_embed_ids;
    fill_cmd.output_embedding_ids = {dist_id}; // Store logits for the last token in this distribution
    handle_fill_block({fill_cmd});
    std::cout << "handle_fill_block completed." << std::endl;

    // 5. Verify the output by sampling the resulting distribution.
    std::cout << "\n[Step 4] Verifying output with handle_sample_top_k..." << std::endl;
    Model::SampleTopKCommand sample_cmd;
    sample_cmd.distribution_id = dist_id;
    sample_cmd.k = 5; // Get top 5 predictions
    auto results = handle_sample_top_k({sample_cmd});

    if (!results.empty()) {
        const auto& result = results[0];
        std::cout << "Successfully retrieved Top-" << result.token_ids.size() << " predicted next tokens:" << std::endl;
        for (size_t i = 0; i < result.token_ids.size(); ++i) {
            std::cout << "  - Token ID: " << result.token_ids[i]
                      << ", Probability: " << result.probabilities[i] << std::endl;
        }
    } else {
        std::cerr << "Test Error: Failed to get sampling results." << std::endl;
    }

    std::cout << "\n--- [END] Test Routine Finished ---\n" << std::endl;
}

// --- New: Public handler methods delegating to PIMPL ---
// These methods are the public API of your Model class. They simply
// forward the calls to the actual implementation in ModelImpl.

void Model::handle_allocate(const std::vector<AllocateCommand>& commands) {
    pimpl->handle_allocate(commands);
}

void Model::handle_deallocate(const std::vector<DeallocateCommand>& commands) {
    pimpl->handle_deallocate(commands);
}

void Model::handle_embed_text(const std::vector<EmbedTextCommand>& commands) {
    pimpl->handle_embed_text(commands);
}

void Model::handle_fill_block(const std::vector<FillBlockCommand>& commands) {
    pimpl->handle_fill_block(commands);
}

void Model::handle_mask_block(const std::vector<MaskBlockCommand>& commands) {
    pimpl->handle_mask_block(commands);
}

void Model::handle_copy_block(const std::vector<CopyBlockCommand>& commands) {
    pimpl->handle_copy_block(commands);
}

void Model::handle_decode_token_distribution(const std::vector<DecodeTokenDistributionCommand>& commands) {
    pimpl->handle_decode_token_distribution(commands);
}

std::vector<Model::SampleTopKResult> Model::handle_sample_top_k(const std::vector<SampleTopKCommand>& commands) {
    return pimpl->handle_sample_top_k(commands);
}