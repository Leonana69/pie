#include "hip/hip_runtime.h"
#include "model.hpp"

// All implementation-specific headers are safely included here
#include "l4ma.cuh"
#include "ztensor.hpp"
#include "common.cuh"
#include <iostream>
#include <set>
#include <memory>
#include <stdexcept>
#include <hip/hip_runtime.h>
#include <vector>
#include <map>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>


// --- Internal data structures ---

// Represents a block in the KV cache on the CPU.
struct Block {
    std::vector<uint32_t> position_ids;
    std::vector<bool> occupancy;

    Block() = default; // Default constructor
    Block(int32_t kv_page_size)
        : position_ids(kv_page_size, 0), occupancy(kv_page_size, false) {}
};

// Represents a text embedding on the CPU.
struct TextEmbed {
    uint32_t token_id;
    uint32_t position_id;
};


// The actual implementation of the Server is hidden in this struct.
struct Model::ModelImpl {
    std::unique_ptr<L4maForCausalLM<__hip_bfloat16>> model;

    // --- State Management ---
    std::map<uint32_t, Block> blocks;
    std::map<uint32_t, TextEmbed> embeds;

    // Storage for results of embedding/inference, analogous to Python's embed_storage
    thrust::device_vector<__hip_bfloat16> embed_storage_p1;
    thrust::device_vector<int32_t> embed_storage_p2;

    // Configuration
    int32_t kv_page_size;
    int32_t dist_size;


    // --- Handler method declarations added to ModelImpl ---
    // These methods contain the core logic and have access to the model pointer.
    void handle_allocate(const std::vector<Model::AllocateCommand>& commands);
    void handle_deallocate(const std::vector<Model::DeallocateCommand>& commands);
    void handle_embed_text(const std::vector<Model::EmbedTextCommand>& commands);
    void handle_fill_block(const std::vector<Model::FillBlockCommand>& commands);
    void handle_mask_block(const std::vector<Model::MaskBlockCommand>& commands);
    void handle_copy_block(const std::vector<Model::CopyBlockCommand>& commands);
    void handle_decode_token_distribution(const std::vector<Model::DecodeTokenDistributionCommand>& commands);
    std::vector<Model::SampleTopKResult> handle_sample_top_k(const std::vector<Model::SampleTopKCommand>& commands);
};

namespace { 

template<typename T>
std::unique_ptr<L4maForCausalLM<T>> load_model_internal(const AppConfig& config, const ModelMetadata& metadata) {
    std::cout << "Instantiating model structure on device..." << std::endl;

    auto model_ptr = std::make_unique<L4maForCausalLM<T>>(metadata.architecture);

    auto params_map = model_ptr->get_parameters();
    std::cout << "Found " << params_map.size() << " parameter tensors in the model structure." << std::endl;

    const auto model_dir = config.cache_dir / config.model_name;
    std::set<std::string> loaded_keys;

    for (const auto& param_file : metadata.parameters) {
        std::filesystem::path weights_path = model_dir / param_file;
        std::cout << "Reading weights from: " << weights_path.string() << std::endl;

        try {
            ztensor::zTensorReader reader(weights_path.string());
            for (const auto& name : reader.list_tensors()) {
                if (params_map.count(name) && !loaded_keys.count(name)) {
                    const auto& info = reader.get_tensor_info(name);
                    thrust::device_vector<T>* target_tensor = params_map[name];

                    if (target_tensor->size() != info.num_elements()) {
                        std::cerr << "    Warning: Shape mismatch for tensor '" << name << "'. ZT: " << info.num_elements() << ", Model: " << target_tensor->size() << ". Skipping." << std::endl;
                        continue;
                    }

                    const T* host_ptr = static_cast<const T*>(reader.get_raw_tensor_pointer(name));
                    if (host_ptr) {
                        hipMemcpy(thrust::raw_pointer_cast(target_tensor->data()), host_ptr, info.size, hipMemcpyHostToDevice);
                        loaded_keys.insert(name);
                    }
                }
            }
        } catch (const std::runtime_error& e) {
            std::cerr << "Warning: Could not read file " << weights_path.string() << ". Error: " << e.what() << std::endl;
        }
    }

    if (params_map.count("lm_head.weight") && params_map.count("model.embed_tokens.weight")) {
        *params_map["lm_head.weight"] = *params_map["model.embed_tokens.weight"];
        loaded_keys.insert("lm_head.weight");
    }
    
    if (loaded_keys.size() != params_map.size()) {
        std::cout << "\nWarning: Mismatch between loaded and expected parameter counts." << std::endl;
        std::cout << "Missing parameters:" << std::endl;
        for (const auto& pair : params_map) {
            if (loaded_keys.find(pair.first) == loaded_keys.end()) {
                std::cout << "  - " << pair.first << std::endl;
            }
        }
    }
    
    std::cout << "\nSuccessfully loaded " << loaded_keys.size() << " expected weights." << std::endl;

    return model_ptr;
}

} // anonymous namespace

// --- New: Placeholder implementations for handler methods ---
// These are the actual implementations within the ModelImpl struct.

void Model::ModelImpl::handle_allocate(const std::vector<Model::AllocateCommand>& commands) {
    std::cout << "  [ModelImpl] handle_allocate called with " << commands.size() << " items." << std::endl;
    for (const auto& cmd : commands) {
        if (cmd.kind == Model::ObjectKind::KV_BLOCK) {
            for (uint32_t i = 0; i < cmd.count; ++i) {
                uint32_t block_id = cmd.object_id_offset + i;
                blocks[block_id] = Block(kv_page_size);
            }
        }
    }
}

void Model::ModelImpl::handle_deallocate(const std::vector<Model::DeallocateCommand>& commands) {
    std::cout << "  [ModelImpl] handle_deallocate called with " << commands.size() << " items." << std::endl;
    // Currently a no-op, as in the Python implementation.
    // Blocks are cleared implicitly when the model is destroyed.
}

void Model::ModelImpl::handle_embed_text(const std::vector<Model::EmbedTextCommand>& commands) {
    std::cout << "  [ModelImpl] handle_embed_text called with " << commands.size() << " items." << std::endl;
    for (const auto& cmd : commands) {
        embeds[cmd.embedding_id] = {cmd.token_id, cmd.position_id};
    }
}

void Model::ModelImpl::handle_fill_block(const std::vector<Model::FillBlockCommand>& commands) {
    std::vector<int32_t> kv_page_indices_host;
    std::vector<int32_t> kv_page_indptr_host = {0};
    std::vector<int32_t> kv_last_page_lens_host;
    std::vector<int32_t> qo_indptr_host = {0};
    std::vector<uint8_t> custom_masks_host; // Use uint8_t for flashinfer
    std::vector<int32_t> mask_indptr_host = {0};

    std::vector<int32_t> kv_batch_indices_host;
    std::vector<int32_t> kv_positions_host;

    std::vector<uint32_t> new_token_ids_host;    // Use uint32_t for model input
    std::vector<uint32_t> new_position_ids_host; // Use uint32_t for model input

    struct OutputEmbedPostproc {
        size_t logit_row_idx;
        uint32_t dest_embed_id;
    };
    std::vector<OutputEmbedPostproc> output_embed_postproc;

    int batch_idx = 0;
    for (const auto& cmd : commands) {
        kv_page_indices_host.insert(kv_page_indices_host.end(), cmd.context_block_ids.begin(), cmd.context_block_ids.end());
        kv_page_indptr_host.push_back(kv_page_indices_host.size());
        kv_last_page_lens_host.push_back(cmd.last_block_len);

        int32_t num_new_tokens = cmd.input_embedding_ids.size();
        qo_indptr_host.push_back(qo_indptr_host.back() + num_new_tokens);

        size_t total_ctx_tokens = (cmd.context_block_ids.empty()) ? 0 :
                                  kv_page_size * (cmd.context_block_ids.size() - 1) + cmd.last_block_len;

        mask_indptr_host.push_back(mask_indptr_host.back() + (num_new_tokens * total_ctx_tokens));

        // --- kv_batch_indices and kv_positions for KV cache update ---
        for (int32_t i = 0; i < num_new_tokens; ++i) {
            kv_batch_indices_host.push_back(batch_idx);
            kv_positions_host.push_back(total_ctx_tokens + i);
        }

        std::vector<uint32_t> inp_pos_ids_for_mask;
        size_t tokens_in_batch_before_cmd = new_token_ids_host.size();

        for (uint32_t embed_id : cmd.input_embedding_ids) {
            auto it = embeds.find(embed_id);
            if (it != embeds.end()) {
                const auto& embed = it->second;
                new_token_ids_host.push_back(embed.token_id);
                new_position_ids_host.push_back(embed.position_id);
                inp_pos_ids_for_mask.push_back(embed.position_id);

                size_t token_abs_pos = total_ctx_tokens + (new_token_ids_host.size() - tokens_in_batch_before_cmd) - 1;
                uint32_t tgt_block_idx = token_abs_pos / kv_page_size;
                uint32_t tgt_block_offset = token_abs_pos % kv_page_size;

                if (tgt_block_idx < cmd.context_block_ids.size()) {
                    uint32_t tgt_block_id = cmd.context_block_ids[tgt_block_idx];
                    auto block_it = blocks.find(tgt_block_id);
                    if (block_it != blocks.end()) {
                        block_it->second.occupancy[tgt_block_offset] = true;
                        block_it->second.position_ids[tgt_block_offset] = embed.position_id;
                    }
                }
            }
        }

        for (size_t i = 0; i < cmd.output_embedding_ids.size(); ++i) {
            size_t logit_row = new_token_ids_host.size() - cmd.output_embedding_ids.size() + i;
            output_embed_postproc.push_back({logit_row, cmd.output_embedding_ids[i]});
        }

        if (total_ctx_tokens > 0) {
            std::vector<uint32_t> ctx_pos_ids;
            std::vector<bool> ctx_occupancy;
            ctx_pos_ids.reserve(total_ctx_tokens);
            ctx_occupancy.reserve(total_ctx_tokens);

            for (size_t i = 0; i < cmd.context_block_ids.size(); ++i) {
                uint32_t block_id = cmd.context_block_ids[i];
                const auto& block = blocks.at(block_id);
                size_t len_to_copy = (i == cmd.context_block_ids.size() - 1) ? cmd.last_block_len : kv_page_size;
                ctx_pos_ids.insert(ctx_pos_ids.end(), block.position_ids.begin(), block.position_ids.begin() + len_to_copy);
                ctx_occupancy.insert(ctx_occupancy.end(), block.occupancy.begin(), block.occupancy.begin() + len_to_copy);
            }

            for (uint32_t inp_pos_id : inp_pos_ids_for_mask) {
                for (size_t j = 0; j < total_ctx_tokens; ++j) {
                    bool causal_mask = ctx_pos_ids[j] <= inp_pos_id;
                    bool valid_mask = ctx_occupancy[j];
                    custom_masks_host.push_back((causal_mask && valid_mask) ? 1 : 0);
                }
            }
        }
        batch_idx++;
    }

    // --- Copy data to device ---
    thrust::device_vector<int32_t> kv_page_indices = kv_page_indices_host;
    thrust::device_vector<int32_t> kv_page_indptr = kv_page_indptr_host;
    thrust::device_vector<int32_t> kv_last_page_lens = kv_last_page_lens_host;
    thrust::device_vector<int32_t> qo_indptr = qo_indptr_host;
    thrust::device_vector<uint8_t> custom_mask = custom_masks_host;
    thrust::device_vector<int32_t> mask_indptr = mask_indptr_host;
    thrust::device_vector<uint32_t> new_token_ids = new_token_ids_host;
    thrust::device_vector<uint32_t> new_position_ids = new_position_ids_host;
    thrust::device_vector<int32_t> kv_batch_indices = kv_batch_indices_host;
    thrust::device_vector<int32_t> kv_positions = kv_positions_host;

    // --- Allocate buffers ---
    size_t num_total_new_tokens = new_token_ids.size();
    if (num_total_new_tokens == 0) return; // Nothing to do

    thrust::device_vector<__hip_bfloat16> logits(num_total_new_tokens * model->get_config().vocab_size);
    

    // using the numbers from flashinfer repo
    thrust::device_vector<char> workspace_buffer_float(128 * 1024 * 1024);
    thrust::device_vector<char> workspace_buffer_int(8 * 1024 * 1024);

    hipStream_t stream = 0; // Use default stream

    

    // --- Model Forward Pass ---
    model->forward(
        logits,
        new_token_ids,
        new_position_ids,
        kv_page_indices,
        kv_page_indptr,
        kv_last_page_lens,
        qo_indptr,
        custom_mask,
        mask_indptr,
        stream,
        workspace_buffer_float,
        workspace_buffer_int,
        kv_page_size,
        kv_batch_indices,
        kv_positions
    );

    if (!output_embed_postproc.empty()) {
        std::vector<size_t> logit_indices_host;
        std::vector<uint32_t> dest_embed_ids_host;
        logit_indices_host.reserve(output_embed_postproc.size());
        dest_embed_ids_host.reserve(output_embed_postproc.size());
        for (const auto& p : output_embed_postproc) {
            logit_indices_host.push_back(p.logit_row_idx);
            dest_embed_ids_host.push_back(p.dest_embed_id);
        }
        thrust::device_vector<size_t> logit_indices_dev = logit_indices_host;
        thrust::device_vector<uint32_t> dest_embed_ids_dev = dest_embed_ids_host;

        topk_scatter(
            logits,
            logit_indices_dev,
            dest_embed_ids_dev,
            model->get_config().vocab_size,
            dist_size,
            embed_storage_p1,
            embed_storage_p2,
            stream
        );
       
    }

}

void Model::ModelImpl::handle_mask_block(const std::vector<Model::MaskBlockCommand>& commands) {
    std::cout << "  [ModelImpl] handle_mask_block called with " << commands.size() << " items." << std::endl;
    for (const auto& cmd : commands) {
        auto it = blocks.find(cmd.block_id);
        if (it != blocks.end()) {
            Block& block = it->second;
            if (block.occupancy.size() == cmd.mask.size()) {
                block.occupancy = cmd.mask;
            } else {
                std::cerr << "Warning: Mask size mismatch for block " << cmd.block_id << std::endl;
            }
        } else {
            std::cerr << "Warning: Block not found for masking: " << cmd.block_id << std::endl;
        }
    }
}

void Model::ModelImpl::handle_copy_block(const std::vector<Model::CopyBlockCommand>& commands) {
    std::cout << "  [ModelImpl] handle_copy_block called with " << commands.size() << " items." << std::endl;
    // TODO: Implement hipMemcpy between different KV cache pages on the device.
    // This requires getting raw pointers to the device vectors for each layer in the KV cache.
}

void Model::ModelImpl::handle_decode_token_distribution(const std::vector<Model::DecodeTokenDistributionCommand>& commands) {
    std::cout << "  [ModelImpl] handle_decode_token_distribution called with " << commands.size() << " items." << std::endl;
    // This is a no-op in the provided python implementation.
    // The logic is integrated into fill_block where top-k results are computed and stored directly.
}

std::vector<Model::SampleTopKResult> Model::ModelImpl::handle_sample_top_k(const std::vector<Model::SampleTopKCommand>& commands) {
    std::cout << "  [ModelImpl] handle_sample_top_k called with " << commands.size() << " items." << std::endl;
    std::vector<Model::SampleTopKResult> results;
    results.reserve(commands.size());

    for (const auto& cmd : commands) {
        Model::SampleTopKResult res;
        
        // Determine the number of elements to copy
        uint32_t k = (cmd.k > 0 && cmd.k < static_cast<uint32_t>(dist_size)) ? cmd.k : dist_size;

        // Create host vectors to hold the results
        thrust::host_vector<__hip_bfloat16> topk_probs_host(k);
        thrust::host_vector<int32_t> topk_tokens_host(k);

        // Copy data from device to host
        hipMemcpy(topk_probs_host.data(), thrust::raw_pointer_cast(embed_storage_p1.data()) + cmd.distribution_id * dist_size, k * sizeof(__hip_bfloat16), hipMemcpyDeviceToHost);
        hipMemcpy(topk_tokens_host.data(), thrust::raw_pointer_cast(embed_storage_p2.data()) + cmd.distribution_id * dist_size, k * sizeof(int32_t), hipMemcpyDeviceToHost);
        
        res.token_ids.assign(topk_tokens_host.begin(), topk_tokens_host.end());
        
        res.probabilities.resize(k);
        for(size_t i = 0; i < k; ++i) {
            res.probabilities[i] = static_cast<float>(topk_probs_host[i]);
        }

        results.push_back(res);
    }
    return results;
}

// --- Public Interface Implementation ---

Model::Model(const AppConfig& config,const ModelMetadata& out_metadata)
    : pimpl(std::make_unique<ModelImpl>()) {
    
    std::cout << "Starting service..." << std::endl;
    // Load the model and store it in the implementation object
    pimpl->model = load_model_internal<__hip_bfloat16>(config, out_metadata);
    std::cout << "Model loaded successfully and is resident on the GPU." << std::endl;

    // initialize kv cache
    pimpl->model->create_kv_device_vectors(config.max_num_kv_pages);

    // Initialize state
    pimpl->kv_page_size = config.kv_page_size;
    pimpl->dist_size = config.dist_size;
    pimpl->embed_storage_p1.resize(config.max_num_embeds * config.dist_size);
    pimpl->embed_storage_p2.resize(config.max_num_embeds * config.dist_size);
}

Model::~Model() = default;

void Model::run() {
    // This would contain the primary execution loop if the model ran continuously.
    // For a request/response server, it can remain empty.
}

// --- New: Public handler methods delegating to PIMPL ---
// These methods are the public API of your Model class. They simply
// forward the calls to the actual implementation in ModelImpl.

void Model::handle_allocate(const std::vector<AllocateCommand>& commands) {
    pimpl->handle_allocate(commands);
}

void Model::handle_deallocate(const std::vector<DeallocateCommand>& commands) {
    pimpl->handle_deallocate(commands);
}

void Model::handle_embed_text(const std::vector<EmbedTextCommand>& commands) {
    pimpl->handle_embed_text(commands);
}

void Model::handle_fill_block(const std::vector<FillBlockCommand>& commands) {
    pimpl->handle_fill_block(commands);
}

void Model::handle_mask_block(const std::vector<MaskBlockCommand>& commands) {
    pimpl->handle_mask_block(commands);
}

void Model::handle_copy_block(const std::vector<CopyBlockCommand>& commands) {
    pimpl->handle_copy_block(commands);
}

void Model::handle_decode_token_distribution(const std::vector<DecodeTokenDistributionCommand>& commands) {
    pimpl->handle_decode_token_distribution(commands);
}

std::vector<Model::SampleTopKResult> Model::handle_sample_top_k(const std::vector<SampleTopKCommand>& commands) {
    return pimpl->handle_sample_top_k(commands);
}