#include "hip/hip_runtime.h"
#include "l4ma.cuh"
#include <hipblas.h>
#include <hipblaslt.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <stdexcept>
#include <cassert>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <flashinfer/activation.cuh>

// Macro for cuBLAS error checking
#define CUBLAS_CHECK(status) \
    do { \
        hipblasStatus_t _status = (status); \
        if (_status != HIPBLAS_STATUS_SUCCESS) { \
            printf("cuBLAS error at %s:%d: %d\n", __FILE__, __LINE__, _status); \
            throw std::runtime_error("cuBLAS error"); \
        } \
    } while (0)


__device__ __forceinline__ float silu(const float& val) { return val / (1.0f + __expf(-val)); }


template <typename T>
void silu_and_mul(
    thrust::device_vector<T>& out,
    const thrust::device_vector<T>& input,
    int num_tokens,
    int d,
    hipStream_t stream,
    bool enable_pdl = false
) {
    T* out_ptr = thrust::raw_pointer_cast(out.data());
    const T* input_ptr = thrust::raw_pointer_cast(input.data());
    uint32_t vec_size = 16 / sizeof(T);
    hipLaunchConfig_t config;
    config.gridDim = num_tokens;
    config.blockDim = std::min(d / vec_size, 1024U);
    config.dynamicSmemBytes = 0;
    config.stream = stream;
    hipLaunchAttribute attrs[1];
    attrs[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
    attrs[0].val.programmaticStreamSerializationAllowed = enable_pdl;
    config.numAttrs = 1;
    config.attrs = attrs;

    auto kernel = flashinfer::activation::act_and_mul_kernel<T, silu>;
    cudaLaunchKernelEx(&config, kernel, out_ptr, input_ptr, d);

}

// Helper for GEMM with bias using cublasLt, supporting float, half, and bf16
inline void gemm_bias_cublasLt(
    hipblasLtHandle_t ltHandle,
    hipStream_t stream,
    const void* A, // [m, k]
    const void* B, // [n, k] (will be transposed)
    const void* bias, // [n] or nullptr
    void* C, // [m, n]
    int m, int n, int k,
    hipDataType dtype, // HIP_R_32F, HIP_R_16F, HIP_R_16BF
    hipblasComputeType_t computeType // HIPBLAS_COMPUTE_32F, HIPBLAS_COMPUTE_16F, CUBLAS_COMPUTE_16BF
) {
    hipblasLtMatmulDesc_t matmulDesc;
    hipblasLtMatrixLayout_t Adesc, Bdesc, Cdesc;
    float alpha = 1.0f, beta = 0.0f;
    hipblasLtEpilogue_t epilogue = bias ? HIPBLASLT_EPILOGUE_BIAS : HIPBLASLT_EPILOGUE_DEFAULT;

    CUBLAS_CHECK(hipblasLtMatmulDescCreate(&matmulDesc, computeType, dtype));
    hipblasOperation_t opA = HIPBLAS_OP_N;
    hipblasOperation_t opB = HIPBLAS_OP_T;
    hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &opA, sizeof(hipblasOperation_t));
    hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &opB, sizeof(hipblasOperation_t));
    hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue));
    if (bias) {
        hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias, sizeof(bias));
    }
    CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&Adesc, dtype, m, k, m));
    CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&Bdesc, dtype, n, k, n));
    CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&Cdesc, dtype, m, n, m));

    CUBLAS_CHECK(hipblasLtMatmul(
        ltHandle,
        matmulDesc,
        &alpha,
        A, Adesc,
        B, Bdesc,
        &beta,
        C, Cdesc,
        C, Cdesc,
        nullptr, nullptr, 0, stream));

    hipblasLtMatmulDescDestroy(matmulDesc);
    hipblasLtMatrixLayoutDestroy(Adesc);
    hipblasLtMatrixLayoutDestroy(Bdesc);
    hipblasLtMatrixLayoutDestroy(Cdesc);
}



// L4maMlp implementation
L4maMlp<float>::L4maMlp(const L4maConfig &config,
        const thrust::device_vector<float> &gate_proj_weights,
        const thrust::device_vector<float> &up_proj_weights,
        const thrust::device_vector<float> &down_proj_weights,
        std::optional<thrust::device_vector<float>> gate_proj_bias,
        std::optional<thrust::device_vector<float>> up_proj_bias,
        std::optional<thrust::device_vector<float>> down_proj_bias)
    : config_(config),
      gate_proj_weights_(gate_proj_weights),
      up_proj_weights_(up_proj_weights),
      down_proj_weights_(down_proj_weights),
      gate_proj_bias_(std::move(gate_proj_bias)),
      up_proj_bias_(std::move(up_proj_bias)),
      down_proj_bias_(std::move(down_proj_bias))
{
    hipblasCreate(&cublas_handle_);
    hipblasLtCreate(&cublaslt_handle_);
}

L4maMlp<float>::~L4maMlp() {
    hipblasDestroy(cublas_handle_);
    hipblasLtDestroy(cublaslt_handle_);
}

void L4maMlp<float>::forward(
    thrust::device_vector<float> &output,
    const thrust::device_vector<float> &x,
    int num_tokens,
    thrust::device_vector<float> &temp_buffer_mlp,
    hipStream_t stream)
{
    // Shapes:
    // x: [num_tokens, hidden_size]
    // gate_proj_weights_: [intermediate_size, hidden_size]
    // up_proj_weights_: [intermediate_size, hidden_size]
    // down_proj_weights_: [hidden_size, intermediate_size]
    // output: [num_tokens, hidden_size]
    // temp_buffer_mlp: must be at least 2 * num_tokens * intermediate_size + num_tokens * hidden_size

    assert(x.size() == num_tokens * config_.hidden_size);
    assert(output.size() == num_tokens * config_.hidden_size);
    int hs = config_.hidden_size;
    int is = config_.intermediate_size;

    const float* x_ptr = thrust::raw_pointer_cast(x.data());
    float* gate_proj_ptr = thrust::raw_pointer_cast(temp_buffer_mlp.data()); // [num_tokens, intermediate_size]
    float* up_proj_ptr = gate_proj_ptr + num_tokens * is; // [num_tokens, intermediate_size]
    float* silu_ptr = up_proj_ptr + num_tokens * is; // [num_tokens, intermediate_size]
    float* mul_ptr = silu_ptr; // reuse silu_ptr for mul result
    float* out_ptr = thrust::raw_pointer_cast(output.data());

    // 1. gate_proj = x * W_g^T (+ b_g)
    gemm_bias_cublasLt(
        cublaslt_handle_, stream,
        x_ptr, thrust::raw_pointer_cast(gate_proj_weights_.data()),
        gate_proj_bias_ ? thrust::raw_pointer_cast(gate_proj_bias_->data()) : nullptr,
        gate_proj_ptr,
        num_tokens, is, hs,
        HIP_R_32F, HIPBLAS_COMPUTE_32F);

    // 2. up_proj = x * W_u^T (+ b_u)
    gemm_bias_cublasLt(
        cublaslt_handle_, stream,
        x_ptr, thrust::raw_pointer_cast(up_proj_weights_.data()),
        up_proj_bias_ ? thrust::raw_pointer_cast(up_proj_bias_->data()) : nullptr,
        up_proj_ptr,
        num_tokens, is, hs,
        HIP_R_32F, HIPBLAS_COMPUTE_32F);

    // 3+4. Fused SiLU activation and elementwise multiply using flashinfer kernel
    silu_and_mul<float>(
        temp_buffer_mlp, // out: silu_ptr (same as mul_ptr)
        temp_buffer_mlp, // input: concat(gate_proj_ptr, up_proj_ptr)
        num_tokens,
        is,
        stream
    );

    // 5. Down projection: output = mul_ptr * W_d^T (+ b_d)
    gemm_bias_cublasLt(
        cublaslt_handle_, stream,
        mul_ptr, thrust::raw_pointer_cast(down_proj_weights_.data()),
        down_proj_bias_ ? thrust::raw_pointer_cast(down_proj_bias_->data()) : nullptr,
        out_ptr,
        num_tokens, hs, is,
        HIP_R_32F, HIPBLAS_COMPUTE_32F);
}
