#include "hip/hip_runtime.h"
#include "common.cuh"
#include <hipblaslt.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <stdexcept>
#include <type_traits>

// Simple macro for checking CUBLAS API calls
#define CUBLAS_CHECK(status)                                    \
    do                                                          \
    {                                                           \
        hipblasStatus_t err = (status);                          \
        if (err != HIPBLAS_STATUS_SUCCESS)                       \
        {                                                       \
            fprintf(stderr, "cuBLAS error at %s:%d, code=%d\n", \
                    __FILE__, __LINE__, err);                   \
            exit(EXIT_FAILURE);                                 \
        }                                                       \
    } while (0)

/**
 * @brief High-performance CUDA kernel for embedding lookup.
 *
 * This version is specialized for uint32_t indices and uses 128-bit
 * vectorized memory operations for maximum bandwidth.
 *
 * @tparam T The base data type (float, __half, etc.).
 */
template <typename T>
__global__ void embedding_lookup_kernel_128bit(T *output,
                                               const T *embedding_matrix,
                                               const uint32_t *indices,
                                               int n,
                                               int hidden_dim_div_16)
{
    // Each block processes one lookup index.
    int idx_n = blockIdx.x;
    if (idx_n >= n)
    {
        return;
    }

    // Use shared memory to broadcast the source row index for the block.
    __shared__ uint32_t source_row_idx;
    if (threadIdx.x == 0)
    {
        source_row_idx = indices[idx_n];
    }
    __syncthreads();

    // Cast pointers to a 128-bit type (float4) to perform 16-byte memory transfers.
    // This is the core optimization, significantly increasing memory throughput.
    const float4 *source_row_ptr = reinterpret_cast<const float4 *>(embedding_matrix) + (long long)source_row_idx * hidden_dim_div_16;
    float4 *dest_row_ptr = reinterpret_cast<float4 *>(output) + (long long)idx_n * hidden_dim_div_16;

    // Use a grid-stride loop for threads to collectively copy the entire row.
    // This ensures that all data is copied regardless of the number of threads per block.
    for (int i = threadIdx.x; i < hidden_dim_div_16; i += blockDim.x)
    {
        dest_row_ptr[i] = source_row_ptr[i];
    }
}

/**
 * @brief Host-side launch function with a Thrust-based API.
 *
 * This function provides a clean, library-style interface using thrust::device_vector
 * and an explicit CUDA stream for asynchronous execution. It assumes uint32_t indices.
 *
 * @tparam T The base data type (float, __half, etc.).
 */
template <typename T>
void embed(
    const thrust::device_vector<T> &embedding,
    const thrust::device_vector<uint32_t> &indices,
    thrust::device_vector<T> *result,
    int embed_width,
    hipStream_t stream)
{
    // --- Input Validation ---
    if (embedding.size() == 0 || indices.size() == 0)
        return;
    if (embedding.size() % embed_width != 0)
    {
        throw std::invalid_argument("Embedding vector size is not divisible by the embed_width.");
    }
    if ((embed_width * sizeof(T)) % 16 != 0)
    {
        throw std::invalid_argument("Total byte size of a slice (embed_width * sizeof(T)) must be a multiple of 16.");
    }

    // --- Prepare Parameters ---
    const int num_indices = indices.size();
    result->resize((long long)num_indices * embed_width);

    const int threads_per_block = 256;
    const int hidden_dim_div_16 = (embed_width * sizeof(T)) / 16;

    dim3 blocks(num_indices);
    dim3 threads(threads_per_block);

    // --- Kernel Launch ---
    embedding_lookup_kernel_128bit<T><<<blocks, threads, 0, stream>>>(
        thrust::raw_pointer_cast(result->data()),
        thrust::raw_pointer_cast(embedding.data()),
        thrust::raw_pointer_cast(indices.data()),
        num_indices,
        hidden_dim_div_16);
}

// --- Explicit Template Instantiations ---
// We explicitly instantiate the templates for the supported types. This forces
// the compiler to generate the code for each of these types, which will then
// be linked against when another file includes embedding.h.

template void embed<float>(
    const thrust::device_vector<float> &,
    const thrust::device_vector<uint32_t> &,
    thrust::device_vector<float> *,
    int, hipStream_t);

template void embed<__half>(
    const thrust::device_vector<__half> &,
    const thrust::device_vector<uint32_t> &,
    thrust::device_vector<__half> *,
    int, hipStream_t);

template void embed<__hip_bfloat16>(
    const thrust::device_vector<__hip_bfloat16> &,
    const thrust::device_vector<uint32_t> &,
    thrust::device_vector<__hip_bfloat16> *,
    int, hipStream_t);

template <typename T>
constexpr hipDataType get_cuda_data_type()
{
    if constexpr (std::is_same_v<T, float>)
    {
        return HIP_R_32F;
    }
    else if constexpr (std::is_same_v<T, __half>)
    {
        return HIP_R_16F;
    }
#if __CUDACC_VER_MAJOR__ >= 11
    else if constexpr (std::is_same_v<T, __hip_bfloat16>)
    {
        return HIP_R_16BF;
    }
#endif
    else if constexpr (std::is_same_v<T, double>)
    {
        return HIP_R_64F;
    }
    else if constexpr (std::is_same_v<T, int8_t>)
    {
        return HIP_R_8I;
    }
    // Add other types here as needed...
    else
    {
        // This will produce a compile-time error if an unsupported type is used.
        static_assert(std::is_same_v<T, void>, "Unsupported data type for gemm_cublasLt_improved");
        return HIP_R_32F; // Dummy return to satisfy compiler
    }
}

// Helper to calculate memory alignment in bytes from a raw pointer
static uint32_t getAlignment(const void *ptr)
{
    uintptr_t address = reinterpret_cast<uintptr_t>(ptr);
    if (address == 0)
        return 256;
    if (address % 256 == 0)
        return 256;
    if (address % 128 == 0)
        return 128;
    if (address % 64 == 0)
        return 64;
    if (address % 32 == 0)
        return 32;
    if (address % 16 == 0)
        return 16;
    if (address % 8 == 0)
        return 8;
    if (address % 4 == 0)
        return 4;
    if (address % 2 == 0)
        return 2;
    return 1;
}

template <typename T>
void gemm_cublasLt(hipblasLtHandle_t ltHandle,
                   hipStream_t stream,
                   const thrust::device_vector<T> &A,
                   const thrust::device_vector<T> &B,
                   const thrust::device_vector<T> *bias,
                   thrust::device_vector<T> &C,
                   int m, int n, int k,
                   thrust::device_vector<char> &workspace,
                   bool transa,
                   bool transb)
{
    if (m <= 0 || n <= 0 || k <= 0)
    {
        return;
    }

    const T *d_A = thrust::raw_pointer_cast(A.data());
    const T *d_B = thrust::raw_pointer_cast(B.data());
    T *d_C = thrust::raw_pointer_cast(C.data());
    const T *d_bias = (bias != nullptr && !bias->empty()) ? thrust::raw_pointer_cast(bias->data()) : nullptr;
    void *d_workspace = thrust::raw_pointer_cast(workspace.data());
    size_t workspaceSize = workspace.size();

    float alpha = 1.0f;
    float beta = (d_bias != nullptr) ? 1.0f : 0.0f;

    hipblasLtMatmulDesc_t matmulDesc = nullptr;
    hipblasLtMatrixLayout_t Adesc = nullptr, Bdesc = nullptr, Cdesc = nullptr;
    hipblasLtMatmulPreference_t preference = nullptr;

    // Use the new helper function to get the data type
    hipDataType cuda_dtype = get_cuda_data_type<T>();
    hipblasComputeType_t compute_type;

    if (cuda_dtype == HIP_R_32F)
    {
        compute_type = HIPBLAS_COMPUTE_32F_FAST_TF32;
    }
    else if (cuda_dtype == HIP_R_16F || cuda_dtype == HIP_R_16BF)
    {
        compute_type = HIPBLAS_COMPUTE_32F;
    }
    else
    {
        compute_type = HIPBLAS_COMPUTE_32F;
    }

    CUBLAS_CHECK(hipblasLtMatmulDescCreate(&matmulDesc, compute_type, HIP_R_32F));
    CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&Adesc, cuda_dtype, transa ? k : m, transa ? m : k, transa ? m : k));
    CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&Bdesc, cuda_dtype, transb ? n : k, transb ? k : n, transb ? k : n));
    CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&Cdesc, cuda_dtype, m, n, n));

    hipblasOperation_t opA = transa ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t opB = transb ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    CUBLAS_CHECK(hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &opA, sizeof(opA)));
    CUBLAS_CHECK(hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &opB, sizeof(opB)));

    hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_DEFAULT;
    if (d_bias != nullptr)
    {
        epilogue = HIPBLASLT_EPILOGUE_BIAS;
        void *non_const_bias = const_cast<void *>(static_cast<const void *>(d_bias));
        CUBLAS_CHECK(hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &non_const_bias, sizeof(non_const_bias)));
    }
    CUBLAS_CHECK(hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue)));

    CUBLAS_CHECK(hipblasLtMatmulPreferenceCreate(&preference));
    CUBLAS_CHECK(hipblasLtMatmulPreferenceSetAttribute(preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize)));

    uint32_t alignA = getAlignment(d_A), alignB = getAlignment(d_B), alignC = getAlignment(d_C);
    CUBLAS_CHECK(hipblasLtMatmulPreferenceSetAttribute(preference, CUBLASLT_MATMUL_PREF_MIN_ALIGNMENT_A_BYTES, &alignA, sizeof(alignA)));
    CUBLAS_CHECK(hipblasLtMatmulPreferenceSetAttribute(preference, CUBLASLT_MATMUL_PREF_MIN_ALIGNMENT_B_BYTES, &alignB, sizeof(alignB)));
    CUBLAS_CHECK(hipblasLtMatmulPreferenceSetAttribute(preference, CUBLASLT_MATMUL_PREF_MIN_ALIGNMENT_C_BYTES, &alignC, sizeof(alignC)));

    int returnedResults = 0;
    hipblasLtMatmulHeuristicResult_t heuristicResult = {};
    CUBLAS_CHECK(hipblasLtMatmulAlgoGetHeuristic(ltHandle, matmulDesc, Adesc, Bdesc, Cdesc, Cdesc, preference, 1, &heuristicResult, &returnedResults));

    if (returnedResults == 0)
    {
        fprintf(stderr, "Error: No suitable cuBLASLt algorithm found!\n");
    }
    else
    {
        CUBLAS_CHECK(hipblasLtMatmul(ltHandle, matmulDesc, &alpha, d_A, Adesc, d_B, Bdesc, &beta,
                                    d_C, Cdesc, d_C, Cdesc,
                                    &heuristicResult.algo, d_workspace, workspaceSize, stream));
    }

    hipblasLtMatmulPreferenceDestroy(preference);
    hipblasLtMatmulDescDestroy(matmulDesc);
    hipblasLtMatrixLayoutDestroy(Adesc);
    hipblasLtMatrixLayoutDestroy(Bdesc);
    hipblasLtMatrixLayoutDestroy(Cdesc);
}

template void gemm_cublasLt(hipblasLtHandle_t,
                            hipStream_t,
                            const thrust::device_vector<__hip_bfloat16> &,
                            const thrust::device_vector<__hip_bfloat16> &,
                            const thrust::device_vector<__hip_bfloat16> *,
                            thrust::device_vector<__hip_bfloat16> &,
                            int, int, int,
                            thrust::device_vector<char> &,
                            bool,
                            bool);

template void gemm_cublasLt(hipblasLtHandle_t,
                            hipStream_t,
                            const thrust::device_vector<float> &,
                            const thrust::device_vector<float> &,
                            const thrust::device_vector<float> *,
                            thrust::device_vector<float> &,
                            int, int, int,
                            thrust::device_vector<char> &,
                            bool,
                            bool);