#include "hip/hip_runtime.h"
#include <cassert>
#include <cmath>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

// Include the implementation unit so the template kernel is visible
#include "../../src/l4ma.cu"

int main() {
    using T = float;
    const int n = 1024;

    T *d_x = nullptr, *d_r = nullptr;
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    CUDA_CHECK(hipMalloc(&d_x, n * sizeof(T)));
    CUDA_CHECK(hipMalloc(&d_r, n * sizeof(T)));

    std::vector<T> hx(n), hr(n);
    for (int i = 0; i < n; ++i) { hx[i] = static_cast<T>(i * 0.25f); hr[i] = static_cast<T>(i * -0.5f); }
    CUDA_CHECK(hipMemcpy(d_x, hx.data(), n * sizeof(T), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_r, hr.data(), n * sizeof(T), hipMemcpyHostToDevice));

    add_residual_kernel<T><<<(n + 255) / 256, 256, 0, stream>>>(d_x, d_r, n);
    CUDA_CHECK(hipStreamSynchronize(stream));

    std::vector<T> out(n);
    CUDA_CHECK(hipMemcpy(out.data(), d_x, n * sizeof(T), hipMemcpyDeviceToHost));

    for (int i = 0; i < n; ++i) {
        T expected = hx[i] + hr[i];
        if (std::fabs(out[i] - expected) > 1e-6f) {
            std::cerr << "Mismatch at " << i << ": got " << out[i] << ", exp " << expected << std::endl;
            return 1;
        }
    }

    CUDA_CHECK(hipFree(d_x));
    CUDA_CHECK(hipFree(d_r));
    CUDA_CHECK(hipStreamDestroy(stream));

    std::cout << "PASS: test_add_residual_full" << std::endl;
    return 0;
}