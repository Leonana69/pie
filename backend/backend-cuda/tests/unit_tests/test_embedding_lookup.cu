#include "hip/hip_runtime.h"
#include <cassert>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

#include "../../src/common.cuh"

int main() {
    using T = float;
    using I = int32_t;

    const int embed_width = 8; // multiple of 4 floats => 32 bytes
    const size_t num_rows = 5;
    const size_t num_indices = 3;

    std::vector<T> h_embedding(num_rows * embed_width);
    for (size_t r = 0; r < num_rows; ++r) {
        for (int c = 0; c < embed_width; ++c) {
            h_embedding[r * embed_width + c] = static_cast<T>(r * 100 + c);
        }
    }
    std::vector<I> h_indices = {3, 1, 4};

    T *d_embedding = nullptr, *d_result = nullptr;
    I *d_indices = nullptr;
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    CUDA_CHECK(hipMalloc(&d_embedding, h_embedding.size() * sizeof(T)));
    CUDA_CHECK(hipMalloc(&d_indices, h_indices.size() * sizeof(I)));
    CUDA_CHECK(hipMalloc(&d_result, num_indices * embed_width * sizeof(T)));

    CUDA_CHECK(hipMemcpy(d_embedding, h_embedding.data(), h_embedding.size() * sizeof(T), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_indices, h_indices.data(), h_indices.size() * sizeof(I), hipMemcpyHostToDevice));

    embed<T,I>(d_embedding, num_rows, d_indices, num_indices, d_result, embed_width, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));

    std::vector<T> h_out(num_indices * embed_width);
    CUDA_CHECK(hipMemcpy(h_out.data(), d_result, h_out.size() * sizeof(T), hipMemcpyDeviceToHost));

    for (size_t i = 0; i < num_indices; ++i) {
        I row = h_indices[i];
        for (int c = 0; c < embed_width; ++c) {
            T expected = static_cast<T>(row * 100 + c);
            if (std::fabs(h_out[i * embed_width + c] - expected) > 1e-6f) {
                std::cerr << "Mismatch at idx " << i << ", col " << c << ": got "
                          << h_out[i * embed_width + c] << ", expected " << expected << std::endl;
                return 1;
            }
        }
    }

    CUDA_CHECK(hipFree(d_embedding));
    CUDA_CHECK(hipFree(d_indices));
    CUDA_CHECK(hipFree(d_result));
    CUDA_CHECK(hipStreamDestroy(stream));

    std::cout << "PASS: test_embedding_lookup" << std::endl;
    return 0;
}