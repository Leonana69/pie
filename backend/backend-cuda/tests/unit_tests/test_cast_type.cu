#include "hip/hip_runtime.h"
#include <cassert>
#include <cmath>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>

#include "../../src/common.cuh"

int main() {
    try {
        const size_t n = 2048;
        std::vector<float> h_input(n);
        for (size_t i = 0; i < n; ++i) {
            h_input[i] = static_cast<float>(std::sin(0.01 * i) * 100.0 + 0.25 * i);
        }

        float* d_in = nullptr;
        __hip_bfloat16* d_bf16 = nullptr;
        float* d_round = nullptr;
        hipStream_t stream;
        CUDA_CHECK(hipStreamCreate(&stream));

        CUDA_CHECK(hipMalloc(&d_in, n * sizeof(float)));
        CUDA_CHECK(hipMalloc(&d_bf16, n * sizeof(__hip_bfloat16)));
        CUDA_CHECK(hipMalloc(&d_round, n * sizeof(float)));

        CUDA_CHECK(hipMemcpy(d_in, h_input.data(), n * sizeof(float), hipMemcpyHostToDevice));

        cast_type<float, __hip_bfloat16>(d_in, d_bf16, n, stream);
        cast_type<__hip_bfloat16, float>(d_bf16, d_round, n, stream);
        CUDA_CHECK(hipStreamSynchronize(stream));

        std::vector<float> h_round(n);
        CUDA_CHECK(hipMemcpy(h_round.data(), d_round, n * sizeof(float), hipMemcpyDeviceToHost));

        // bfloat16 has ~7-bit mantissa => expect some quantization. Use relative tolerance.
        int num_bad = 0;
        for (size_t i = 0; i < n; ++i) {
            float a = h_input[i];
            float b = h_round[i];
            float denom = std::max(1.0f, std::fabs(a));
            if (std::fabs(a - b) / denom > 1e-2f) {
                ++num_bad;
                if (num_bad < 10) {
                    std::cerr << "Mismatch at " << i << ": a=" << a << ", b=" << b << std::endl;
                }
            }
        }
        if (num_bad > 0) {
            std::cerr << "Total mismatches: " << num_bad << std::endl;
            return 1;
        }

        CUDA_CHECK(hipFree(d_in));
        CUDA_CHECK(hipFree(d_bf16));
        CUDA_CHECK(hipFree(d_round));
        CUDA_CHECK(hipStreamDestroy(stream));

        std::cout << "PASS: test_cast_type (float <-> bf16 round-trip)" << std::endl;
        return 0;
    } catch (const std::exception& ex) {
        std::cerr << "Exception: " << ex.what() << std::endl;
        return 2;
    }
}